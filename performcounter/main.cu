#include "hip/hip_runtime.h"
#include <nvml.h>
#include <stdlib.h>
#include <stdio.h>
#include <percounter.h>

int main(int argc, char *argv[]){

//init
    if(init() == 0){
		printf("nvml init succ.\n");
	} else {
		printf("nvml init err.\n");
		return 1;
	}
    
//For Variable and malloc
	unsigned int length = 100;
    nvmlDevice_t device;
	
	char *devicename = (char*)malloc(sizeof(char)*length);
	
	unsigned int maxLinkGen;
	
	unsigned int maxLinkWidth;
	
	unsigned int currLinkGen;
	
	unsigned int currLinkWidth;

    unsigned int pciethroughput;	
	
	unsigned int pcieReplayCount;
	
	
	unsigned int currSmClock;
	
	unsigned int currMemClock;
	
	unsigned int smclockMHz;
	
	unsigned int memclockMHz;
	
	unsigned int maxSmClock;
	
	unsigned int maxMemClock;
	
	
	unsigned int currsmclo;
	
	unsigned int currmemclo;
	
	unsigned int appsmclo;
	
	unsigned int appmemclo;
	
	unsigned int defsmclo;
	
	unsigned int defmemclo;
	
	
	
	unsigned int temp;
	
	unsigned int fanspeed;
	
	unsigned int powerusage;
	
	
	
	unsigned long long volatileEccCountsTotalSingle;
	unsigned long long aggregateEccCountsTotalSingle;
	
	unsigned long long volatileEccCountsTotalDouble;
	unsigned long long aggregateEccCountsTotalDouble;
	
//	unsigned long long energyConsumption;
	
	nvmlMemory_t *memory = (nvmlMemory_t*)malloc(sizeof(nvmlMemory_t));
	
	nvmlPstates_t *perState = (nvmlPstates_t*)mallco(sizeof(nvmlPstates_t));
	
	int perforState;
	
	nvmlPstates_t *powState = (nvmlPstates_t*)mallco(sizeof(nvmlPstates_t));
	
	int powerState;
	
	unsigned long long volatileSingleBitErrorL1CacheCount;
	unsigned long long aggregateSingleBitErrorL1CacheCount;
	
	unsigned long long volatileDoubleBitErrorL1CacheCount;
	unsigned long long aggregateDoubleBitErrorL1CacheCount;
	
	unsigned long long volatileSingleBitErrorL2CacheCount;
	unsigned long long aggregateSingleBitErrorL2CacheCount;
	
	unsigned long long volatileDoubleBitErrorL2CacheCount;
	unsigned long long aggregateDoubleBitErrorL2CacheCount;
	
	unsigned long long volatileSingleBitErrorDeviceMemoryCount;
	unsigned long long aggregateSingleBitErrorDeviceMemoryCount;
	
	unsigned long long volatileDoubleBitErrorDeviceMemoryCount;
	unsigned long long aggregateDoubleBitErrorDeviceMemoryCount;
	
	unsigned long long volatileSingleBitErrorRegisterFileCount;
	unsigned long long aggregateSingleBitErrorRegisterFileCount;
	
	unsigned long long volatileDoubleBitErrorRegisterFileCount;
	unsigned long long aggregateDoubleBitErrorRegisterFileCount;
	
	unsigned long long volatileSingleBitErrorTextureMemoryCount;
	unsigned long long aggregateSingleBitErrorTextureMemoryCount;
	
	unsigned long long volatileDoubleBitErrorTextureMemoryCount;
	unsigned long long aggregateDoubleBitErrorTextureMemoryCount;
	
	//unsigned long long volatileSingleBitErrorTextureShmCount;
	//unsigned long long aggregateSingleBitErrorTextureShmCount;
	
	//unsigned long long volatileDoubleBitErrorTextureShmCount;
	//unsigned long long aggregateDoubleBitErrorTextureShmCount;
	
	//unsigned long long volatileSingleBitErrorCBUCount;
	//unsigned long long aggregateSingleBitErrorCBUCount;
	
	//unsigned long long volatileDoubleBitErrorCBUCount;
	//unsigned long long aggregateDoubleBitErrorCBUCount;
	
	nvmlUtilization_t *deviceUtilization = (nvmlUtilization_t*)malloc(sizeof(nvmlUtilization_t));
	
	nvmlProcessInfo_t *infos = (nvmlProcessInfo_t*)malloc(sizeof(nvmlProcessInfo_t));
	
	nvmlBAR1Memory_t *bar1Memory = (nvmlBAR1Memory_t*)malloc(sizeof(nvmlBAR1Memory_t));
	


// For Test

    printf("the clock count is %d.\n",NVML_CLOCK_COUNT);
    printf("the graphics clock is %d.\n",NVML_CLOCK_GRAPHICS);
    printf("the SM clock is %d.\n",NVML_CLOCK_SM);
    printf("the Memory clock is %d.\n",NVML_CLOCK_MEM);
   // printf("the Video clock is %d.\n",NVML_CLOCK_VIDEO);    

//  For getDevicehandle()

    if (getDevicehandle(&device) == 0){
        printf("get device handle success.\n");
		nvmlDeviceGetName(device,devicename,length);
		printf("the name of device 0 is %s.\n",devicename);
	} else {
	    printf("get device handle error.\n");
	    return 1;
	}
	
//For getMaxPcieLinkGeneration
    if(getMaxPcieLinkGeneration(device, &maxLinkGen) == 0){
	    printf("max pcie link generation is %u.\n", maxLinkGen);
	} else {
	    printf("get max pcie link generation err.\n");
	}
	
//For getMaxPcieLinkWidth
    if(getMaxPcieLinkWidth(device, &maxLinkWidth) == 0){
	    printf("max pcie link width is %u.\n", maxLinkWidth);
	} else {
	    printf("get max pcie link width err.\n");
	}

//For getCurrPcieLinkGeneration
    if(getCurrPcieLinkGeneration(device, &currLinkGen) == 0){
	    printf("current pcie link generation is %u.\n", currLinkGen);
	} else {
	
	}

//  For getCurrPcieLinkWidth()
    if(getCurrPcieLinkWidth(device, &currLinkWidth) == 0){
		printf("current pcie link width is %u.\n",currLinkWidth);
	} else {
		printf("get current pcie link width err.\n");

	}

//  For getPcieThroughput()
 //not support
 /*   
	if(getPcieThroughput(device, &pciethroughput) == 0){
		printf("the pcie throughput is %u.\n", pciethroughput);
	} else {
		printf("get pcie throughput error.\n");

	}
*/

//For getPcieReplayCounter()

    if(getPcieReplayCounter(device, &pcieReplayCount) == 0){
	    printf("the pcie replay counter is %u.\n", pcieReplayCount);
	} else {
	    printf("get pcie replay counter error.\n");
	}



//For getCurrSMClockInfo
    if(getCurrSMClockInfo(device, &currSmClock) == 0){
	    printf("the current SM clock speed is %u.\n", currSmClock);
	} else {
	    printf("get current SM clock speed error.\n");
	}

//For getCurrMEMClockInfo
    if (getCurrMEMClockInfo(device, &currMemClock) == 0){
	    printf("the current MEM clock speed is %u.\n", currMemClock);
	} else {
	    printf("get current MEM clock speed error.\n");
	}

//For getSMApplicationsClock
    if(getSMApplicationsClock(device, &smclockMHz) == 0){
	    printf("the application SM clock is %u.\n", smclockMHz);
	} else {
	    printf("get application SM clock error.\n");
	}

//For GetMEMApplicationsClock
    if(GetMEMApplicationsClock(device, &memclockMHz) == 0){
	    printf("the application MEM clock is %u.\n", memclockMHz);
	} else {
	    printf("get application MEM clock error.\n");
	}

//For getMaxSMClockInfo
    if(getMaxSMClockInfo(device, &maxSmClock) == 0){
	    printf("the max SM clock speed is %u.\n", maxSmClock);
	} else {
	    printf("get max SM clock speed error.\n");
	}

//For getMaxMEMClockInfo
    if (getMaxMEMClockInfo(device, &maxMemClock) == 0){
	    printf("the max MEM clock speed is %u.\n", maxMemClock);
	} else {
	    printf("get max MEM clock speed error.\n");
	}


//
    if(getCurrSMClock(device, &currsmclo) == 0){
	    printf("the current SM clock speed is %u.\n", currsmclo);
	} else {
	    printf("get the current SM clock speed error.\n");
	}

//
    if(getCurrMEMClock(device, &currmemclo) ==0){
	    printf("the current MEM clock speed is %u.\n", currmemclo);
	} else {
	    printf("get the current MEM clock speed error.\n");
	}

//
    if(getAppSMClock(device, &appsmclo) == 0){
	    printf("the application SM clock speed is %u.\n", appsmclo);
	} else {
	    printf("get the application SM clock speed error.\n");
	}

//
    if(getAppMEMClock(device, &appmemclo) == 0){
	    printf("the application MEM clock speed is %u.\n", appmemclo);
	} else {
	    printf("get the application MEM clock speed error.\n");
	}

//
    if (getDefSMClock(device, &defsmclo) == 0){
	    printf("the default SM clock speed is %u.\n", defsmclo);
	} else {
	    printf("get the default SM clock speed error.\n");
	}

//
    if(getDefMEMClock(device, &defmemclo) == 0){
	    printf("the default MEM clock speed is %u.\n", defmemclo);
	} else {
	    printf("get the default MEM clock speed error.\n");
	}


//For getTemperature
    if(getTemperature(device, &temp) == 0){
	    printf("the GPU temperature is %u.\n", temp);
	} else {
	    printf("get the GPU temperature error.\n");
	}




//For getFanSpeed()
//not support
/*
    if(getFanSpeed(device, &fanspeed) == 0){
		printf("the fan speed is %u.\n", fanspeed);
	} else {
		printf("get fan speed error.\n");
	}
*/
//For getPowerUsage()

    if(getPowerUsage(device, &powerusage) == 0){
	    printf("the power usage is %u.\n", powerusage);
	} else {
	    printf("get power usage error.\n");
	}

//For getTotalEnergyConsumption()
//volta or newer
/*
    if(getTotalEnergyConsumption(device, &energyConsumption) == 0){
	    printf("the total energy consumption is %lld.\n", energyConsumption);
	} else {
	    printf("get total energy consumption error.\n");
	}
*/

//For getMemoryInfo()
    if(getMemoryInfo(device, memory) == 0){
	    printf("the total memory is %lld.\n", memory->total);
		printf("the free memory is %lld.\n", memory->free);
		printf("the used memory is %lld.\n", memory->used);
    
    } else {
	    printf("get memory info error.\n");
	}


//For getPerformanceState
    if(getPerformanceState(device, perState, perforState) == 0){
	    printf("the performance state is %d.\n",perforState);
	} else {
	    printf("get the performance state error.\n");
	}

//For getPowerState
    if(getPowerState(device, powState, powerState) == 0){
	    printf("the power state is %d.\n",powerState);
	} else {
	    printf("get the power state error.\n");
	}




//For getTotalVolatileSingleBitEccErrors()
    if(getTotalVolatileSingleBitEccErrors(device, &volatileEccCountsTotalSingle) == 0){
	    printf("the number of volatile total single bit ecc error is %lld.\n", volatileEccCountsTotalSingle);
	} else {
	    printf("get volatile total single bit ecc error failed.\n");
	}

//For getTotalAggregateSingleBitEccErrors()
    if(getTotalAggregateSingleBitEccErrors(device, &aggregateEccCountsTotalSingle) == 0){
	    printf("the number of aggregate total single bit ecc error is %lld.\n", aggregateEccCountsTotalSingle);
	} else {
	    printf("get aggregate total single bit ecc error failed.\n");
	}

//For getTotalVolatileDoubleBitEccErrors()
    if(getTotalVolatileDoubleBitEccErrors(device, &volatileEccCountsTotalDouble) == 0){
	    printf("the number of volatile total double bit ecc error is %lld.\n", volatileEccCountsTotalDouble);
	} else {
	    printf("get volatile total double bit ecc error failed.\n");
	}

//For getTotalAggregateDoubleBitEccErrors()
    if(getTotalAggregateDoubleBitEccErrors(device, &aggregateEccCountsTotalDouble) == 0){
	    printf("the number of aggregate total double bit ecc error is %lld.\n", aggregateEccCountsTotalDouble);
	} else {
	    printf("get aggregate total double bit ecc error failed.\n");
	}

//For getMemoryVolatileSingleBitErrorCounterL1Cache()
    if(getMemoryVolatileSingleBitErrorCounterL1Cache(device, &volatileSingleBitErrorL1CacheCount) == 0){
	    printf("the number of volatile single bit ecc error in L1 Cache is %lld.\n", volatileSingleBitErrorL1CacheCount);
	} else {
	    printf("get volatile single bit ecc error in L1 Cache failed.\n");
	}
	
//For getMemoryAggregateSingleBitErrorCounterL1Cache()
    if(getMemoryAggregateSingleBitErrorCounterL1Cache(device, &aggregateSingleBitErrorL1CacheCount) == 0){
	    printf("the number of aggregate single bit ecc error in L1 Cache is %lld.\n", aggregateSingleBitErrorL1CacheCount);
		
	} else {
	    printf("get aggregate single bit ecc error in L1 Cache failed.\n");
	}

//For getMemoryVolatileDoubleBitErrorCounterL1Cache()
    if(getMemoryVolatileDoubleBitErrorCounterL1Cache(device, &volatileDoubleBitErrorL1CacheCount) == 0){
	    printf("the number of volatile double bit ecc error in L1 Cache is %lld.\n", volatileDoubleBitErrorL1CacheCount);
	} else {
	    printf("get volatile double bit ecc error in L1 Cache failed.\n");
	}

//For getMemoryAggregateDoubleBitErrorCounterL1Cache()
    if(getMemoryAggregateDoubleBitErrorCounterL1Cache(device, &aggregateDoubleBitErrorL1CacheCount) == 0){
	    printf("the number of aggregate double bit ecc error in L1 Cache is %lld.\n", aggregateDoubleBitErrorL1CacheCount);
	} else {
	    printf("get aggregate double bit ecc error in L1 Cache failed.\n");
	}
	
//For getMemoryVolatileSingleBitErrorCounterL2Cache()
    if(getMemoryVolatileSingleBitErrorCounterL2Cache(device, &volatileSingleBitErrorL2CacheCount) == 0){
	    printf("the number of volatile single bit ecc error in L2 Cache is %lld.\n", volatileSingleBitErrorL2CacheCount);
	} else {
	    printf("get volatile single bit ecc error in L2 Cache failed.\n");
	}
	
//For getMemoryAggregateSingleBitErrorCounterL2Cache()
    if(getMemoryAggregateSingleBitErrorCounterL2Cache(device, &aggregateSingleBitErrorL2CacheCount) == 0){
	    printf("the aggregate number of single bit ecc error in L2 Cache is %lld.\n", aggregateSingleBitErrorL2CacheCount);
	} else {
	    printf("get aggregate single bit ecc error in L2 Cache failed.\n");
	}
	
//For getMemoryVolatileDoubleBitErrorCounterL2Cache()
    if(getMemoryVolatileDoubleBitErrorCounterL2Cache(device, &volatileDoubleBitErrorL2CacheCount) == 0){
	    printf("the number of volatile double bit ecc error in L2 Cache is %lld.\n", volatileDoubleBitErrorL2CacheCount);
	} else {
	    printf("get volatile double bit ecc error in L2 Cache failed.\n");
	}
	
//For getMemoryAggregateDoubleBitErrorCounterL2Cache()
    if(getMemoryAggregateDoubleBitErrorCounterL2Cache(device, &aggregateDoubleBitErrorL2CacheCount) == 0){
	    printf("the number of aggregate double bit ecc error in L2 Cache is %lld.\n", aggregateDoubleBitErrorL2CacheCount);
	} else {
	    printf("get aggregate double bit ecc error in L2 Cache failed.\n");
	}
	
//For getMemoryVolatileSingleBitErrorCounterDeviceMemory()
    if(getMemoryVolatileSingleBitErrorCounterDeviceMemory(device, &volatileSingleBitErrorDeviceMemoryCount) == 0){
	    printf("the number of volatile single bit ecc error in Device Memory is %lld.\n", volatileSingleBitErrorDeviceMemoryCount);
	} else {
	    printf("get volatile single bit ecc error in Device Memory failed.\n");
	}
//For getMemoryAggregateSingleBitErrorCounterDeviceMemory()
    if(getMemoryAggregateSingleBitErrorCounterDeviceMemory(device, &aggregateSingleBitErrorDeviceMemoryCount) == 0){
	    printf("the number of aggregate single bit ecc error in Device Memory is %lld.\n", aggregateSingleBitErrorDeviceMemoryCount);
	} else {
	    printf("get aggregate single bit ecc error in Device Memory failed.\n");
	}
	
//For getMemoryVolatileDoubleBitErrorCounterDeviceMemory()
    if(getMemoryVolatileDoubleBitErrorCounterDeviceMemory(device, &volatileDoubleBitErrorDeviceMemoryCount) == 0){
	    printf("the number of volatile double bit ecc error in Device Memory is %lld.\n", volatileDoubleBitErrorDeviceMemoryCount);
	} else {
	    printf("get volatile double bit ecc error in Device Memory failed.\n");
	}
//For getMemoryAggregateDoubleBitErrorCounterDeviceMemory()
    if(getMemoryAggregateDoubleBitErrorCounterDeviceMemory(device, &aggregateDoubleBitErrorDeviceMemoryCount) == 0){
	    printf("the number of aggregate double bit ecc error in Device Memory is %lld.\n", aggregateDoubleBitErrorDeviceMemoryCount);
	} else {
	    printf("get aggregate double bit ecc error in Device Memory failed.\n");
	}
	
//For getMemoryVolatileSingleBitErrorCounterRegisterFile()
    if(getMemoryVolatileSingleBitErrorCounterRegisterFile(device, &volatileSingleBitErrorRegisterFileCount) == 0){
	    printf("the number of volatile single bit ecc error in Register File is %lld.\n", volatileSingleBitErrorRegisterFileCount);
	} else {
	    printf("get volatile single bit ecc error in Register File failed.\n");
	}
//For getMemoryAggregateSingleBitErrorCounterRegisterFile()
    if(getMemoryAggregateSingleBitErrorCounterRegisterFile(device, &aggregateSingleBitErrorRegisterFileCount) == 0){
	    printf("the number of aggregate single bit ecc error in Register File is %lld.\n", aggregateSingleBitErrorRegisterFileCount);
	} else {
	    printf("get aggregate single bit ecc error in Register File failed.\n");
	}
	
//For getMemoryVolatileDoubleBitErrorCounterRegisterFile()
    if(getMemoryVolatileDoubleBitErrorCounterRegisterFile(device, &volatileDoubleBitErrorRegisterFileCount) == 0){
	    printf("the number of volatile double bit ecc error in Register File is %lld.\n", volatileDoubleBitErrorRegisterFileCount);
	} else {
	    printf("get volatile double bit ecc error in Register File failed.\n");
	}

//For getMemoryAggregateDoubleBitErrorCounterRegisterFile()
    if(getMemoryAggregateDoubleBitErrorCounterRegisterFile(device, &aggregateDoubleBitErrorRegisterFileCount) == 0){
	    printf("the number of aggregate double bit ecc error in Register File is %lld.\n", aggregateDoubleBitErrorRegisterFileCount);
	} else {
	    printf("get aggregate double bit ecc error in Register File failed.\n");
	}
	
//For getMemoryVolatileSingleBitErrorCounterTextureMemory()
    if(getMemoryVolatileSingleBitErrorCounterTextureMemory(device, &volatileSingleBitErrorTextureMemoryCount) == 0){
	    printf("the number of volatile single bit ecc error in Texture Memory is %lld.\n", volatileSingleBitErrorTextureMemoryCount);
	} else {
	    printf("get volatile single bit ecc error in Texture Memory failed.\n");
	}
	
//For getMemoryAggregateSingleBitErrorCounterTextureMemory()
    if(getMemoryAggregateSingleBitErrorCounterTextureMemory(device, &aggregateSingleBitErrorTextureMemoryCount) == 0){
	    printf("the number of aggregate single bit ecc error in Texture Memory is %lld.\n", aggregateSingleBitErrorTextureMemoryCount);
	} else {
	    printf("get aggregate single bit ecc error in Texture Memory failed.\n");
	}
	
//For getMemoryVolatileDoubleBitErrorCounterTextureMemory()
    if(getMemoryVolatileDoubleBitErrorCounterTextureMemory(device, &volatileDoubleBitErrorTextureMemoryCount) == 0){
	    printf("the number of volatile double bit ecc error in Texture Memory is %lld.\n", volatileDoubleBitErrorTextureMemoryCount);
	} else {
	    printf("get volatile double bit ecc error in Texture Memory failed.\n");
	}
	
//For getMemoryAggregateDoubleBitErrorCounterTextureMemory()
    if(getMemoryAggregateDoubleBitErrorCounterTextureMemory(device, &aggregateDoubleBitErrorTextureMemoryCount) == 0){
	    printf("the number of aggregate double bit ecc error in Texture Memory is %lld.\n", aggregateDoubleBitErrorTextureMemoryCount);
	} else {
	    printf("get aggregate double bit ecc error in Texture Memory failed.\n");
	}
	
	
	
	
	




//For getMemorySingleBitErrorCounterTextureShm()
// not support
/*
    if(getMemorySingleBitErrorCounterTextureShm(device, &singleBitErrorTextureShmCount) == 0){
	    printf("the number of single bit ecc error in Shared memory is %lld.\n", singleBitErrorTextureShmCount);
	} else {
	    printf("get single bit ecc error in Shared memory failed.\n");
	}

//For getMemoryDoubleBitErrorCounterTextureShm()
    if(getMemoryDoubleBitErrorCounterTextureShm(device, &doubleBitErrorTextureShmCount) == 0){
	    printf("the number of double bit ecc error in Shared memory is %lld.\n", doubleBitErrorTextureShmCount);
	} else {
	    printf("get double bit ecc error in Shared memory failed.\n");
	}

//For getMemorySingleBitErrorCounterCBU()
    if(getMemorySingleBitErrorCounterCBU(device, &singleBitErrorCBUCount) == 0){
	    printf("the number of single bit ecc error in CBU is %lld.\n", singleBitErrorCBUCount);
	} else {
	    printf("get single bit ecc error in CBU failed.\n");
	}

//For getMemoryDoubleBitErrorCounterCBU()
    if(getMemoryDoubleBitErrorCounterCBU(device, &doubleBitErrorCBUCount) == 0){
	    printf("the number of double bit ecc error in CBU is %lld.\n", doubleBitErrorCBUCount);
	} else {
	    printf("get double bit ecc error in CBU failed.\n");
	}
*/




//int getUtilizationRates()
    if(getUtilizationRates(device, deviceUtilization) == 0){
	    printf("the Utilization of GPU is %u.\n", deviceUtilization->gpu);
	    printf("the Utilization of Memory is %u.\n", deviceUtilization->memory);
	} else {
	    printf("get device's UtilizationRate failed.\n");
	}


//int getEncoderUtilization()




//int getDecoderUtilization()


//int getComputeRunningProcesses()
    if(getComputeRunningProcesses(device, infos) == 0){
	    printf("Process ID is %u.\n", infos->pid);
		printf("used Gpu Memory is %lld.\n",infos->usedGpuMemory);
	} else {
	    printf("get Compute Running Processes failed.\n");
	}



//int getBAR1MemoryInfo()
    if(getBAR1MemoryInfo(device, bar1Memory) == 0){
	    printf("bar1 total memory is %lld.\n", bar1Memory->bar1Total);
		printf("bar1 free memory is %lld.\n", bar1Memory->bar1Free);
		printf("bar1 used memory is %lld.\n", bar1Memory->bar1Used);
	} else {
	    printf("get bar1 memory infos failed.\n");
	}


//int getViolationStatus()

//shutdown
    if(shutdown() == 0){
		printf("nvml shutdown succ.\n");
	} else {
		printf("nvml shutdown err.\n");
		return 1;
	}
	
// free
   free(devicename);
   free(memory);
   free(perState);
   free(powState);
   free(deviceUtilization);
   free(infos);
   free(bar1Memory);
return 0;	

}



