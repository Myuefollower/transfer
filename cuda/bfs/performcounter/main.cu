#include "hip/hip_runtime.h"
#include <nvml.h>
#include <stdlib.h>
#include <stdio.h>
#include <percounter.h>

int main(int argc, char *argv[]){

//init
    if(init() == 0){
		printf("nvml init succ.\n");
	} else {
		printf("nvml init err.\n");
		return 1;
	}
    
//For Variable and malloc
	unsigned int length = 100;
    nvmlDevice_t device;
	
	char *devicename = (char*)malloc(sizeof(char)*length);
	
	unsigned int linkwid;

    unsigned int pciethroughput;	
	
	unsigned int pcieReplayCount;
	
	unsigned int fanspeed;
	
	unsigned int powerusage;
	
	unsigned long long eccCountsTotalSingle;
	
	unsigned long long eccCountsTotalDouble;
	
//	unsigned long long energyConsumption;
	
	nvmlMemory_t *memory = (nvmlMemory_t*)malloc(sizeof(nvmlMemory_t));
	
	unsigned long long singleBitErrorL1CacheCount;
	
	unsigned long long doubleBitErrorL1CacheCount;
	
	unsigned long long singleBitErrorL2CacheCount;
	
	unsigned long long doubleBitErrorL2CacheCount;
	
	unsigned long long singleBitErrorDeviceMemoryCount;
	
	unsigned long long doubleBitErrorDeviceMemoryCount;
	
	unsigned long long singleBitErrorRegisterFileCount;
	
	unsigned long long doubleBitErrorRegisterFileCount;
	
	unsigned long long singleBitErrorTextureMemoryCount;
	
	unsigned long long doubleBitErrorTextureMemoryCount;
	
	unsigned long long singleBitErrorTextureShmCount;
	
	unsigned long long doubleBitErrorTextureShmCount;
	
	unsigned long long singleBitErrorCBUCount;
	
	unsigned long long doubleBitErrorCBUCount;
	
	nvmlUtilization_t *deviceUtilization = (nvmlUtilization_t*)malloc(sizeof(nvmlUtilization_t));
	
	nvmlProcessInfo_t *infos = (nvmlProcessInfo_t*)malloc(sizeof(nvmlProcessInfo_t));
	
	nvmlBAR1Memory_t *bar1Memory = (nvmlBAR1Memory_t*)malloc(sizeof(nvmlBAR1Memory_t));
	


// For Test

    printf("the clock count is %d.\n",NVML_CLOCK_COUNT);
    printf("the graphics clock is %d.\n",NVML_CLOCK_GRAPHICS);
    printf("the SM clock is %d.\n",NVML_CLOCK_SM);
    printf("the Memory clock is %d.\n",NVML_CLOCK_MEM);
   // printf("the Video clock is %d.\n",NVML_CLOCK_VIDEO);    
//  For getDevicehandle()

    if (getDevicehandle(&device) == 0){
        printf("get device handle success.\n");
		nvmlDeviceGetName(device,devicename,length);
		printf("the name of device 0 is %s.\n",devicename);
	} else {
	    printf("get device handle error.\n");
	    return 1;
	}

//  For getCurrPcieLinkWidth()

    if(getCurrPcieLinkWidth(device, &linkwid) == 0){
		printf("current pcie link width is %u.\n",linkwid);
	} else {
		printf("get current pcie link width err.\n");

	}

//  For getPcieThroughput()
 //not support
 /*   
	if(getPcieThroughput(device, &pciethroughput) == 0){
		printf("the pcie throughput is %u.\n", pciethroughput);
	} else {
		printf("get pcie throughput error.\n");

	}
*/
//For getPcieReplayCounter()

    if(getPcieReplayCounter(device, &pcieReplayCount) == 0){
	    printf("the pcie replay counter is %u.\n",pcieReplayCount);
	} else {
	    printf("get pcie replay counter error.\n");
	}

//For getFanSpeed()
//not support
/*
    if(getFanSpeed(device, &fanspeed) == 0){
		printf("the fan speed is %u.\n", fanspeed);
	} else {
		printf("get fan speed error.\n");
	}
*/
//For getPowerUsage()

    if(getPowerUsage(device, &powerusage) == 0){
	    printf("the power usage is %u.\n", powerusage);
	} else {
	    printf("get power usage error.\n");
	}

//For getTotalEnergyConsumption()
//volta or newer
/*
    if(getTotalEnergyConsumption(device, &energyConsumption) == 0){
	    printf("the total energy consumption is %lld.\n", energyConsumption);
	} else {
	    printf("get total energy consumption error.\n");
	}
*/

//For getMemoryInfo()
    if(getMemoryInfo(device, memory) == 0){
	    printf("the total memory is %lld.\n", memory->total);
		printf("the free memory is %lld.\n", memory->free);
		printf("the used memory is %lld.\n", memory->used);
    
    } else {
	    printf("get memory info error.\n");
	}


//For getTotalSingleBitEccErrors()
    if(getTotalSingleBitEccErrors(device, &eccCountsTotalSingle) == 0){
	    printf("the number of total single bit ecc error is %lld.\n", eccCountsTotalSingle);
	} else {
	    printf("get total single bit ecc error failed.\n");
	}



//For getTotalDoubleBitEccErrors()
    if(getTotalDoubleBitEccErrors(device, &eccCountsTotalDouble) == 0){
	    printf("the number of total double bit ecc error is %lld.\n", eccCountsTotalDouble);
	} else {
	    printf("get total double bit ecc error failed.\n");
	}



//For getMemorySingleBitErrorCounterL1Cache()
    if(getMemorySingleBitErrorCounterL1Cache(device, &singleBitErrorL1CacheCount) == 0){
	    printf("the number of single bit ecc error in L1 Cache is %lld.\n", singleBitErrorL1CacheCount);
		
	} else {
	    printf("get single bit ecc error in L1 Cache failed.\n");
	}


//For getMemoryDoubleBitErrorCounterL1Cache()
    if(getMemoryDoubleBitErrorCounterL1Cache(device, &doubleBitErrorL1CacheCount) == 0){
	    printf("the number of double bit ecc error in L1 Cache is %lld.\n", eccCountsTotalDouble);
	} else {
	    printf("get double bit ecc error in L1 Cache failed.\n");
	}

//For getMemorySingleBitErrorCounterL2Cache()
    if(getMemorySingleBitErrorCounterL2Cache(device, &singleBitErrorL2CacheCount) == 0){
	    printf("the number of single bit ecc error in L2 Cache is %lld.\n", singleBitErrorL2CacheCount);
	} else {
	    printf("get single bit ecc error in L2 Cache failed.\n");
	}

//For getMemoryDoubleBitErrorCounterL2Cache()
    if(getMemoryDoubleBitErrorCounterL2Cache(device, &doubleBitErrorL2CacheCount) == 0){
	    printf("the number of double bit ecc error in L2 Cache is %lld.\n", doubleBitErrorL2CacheCount);
	} else {
	    printf("get double bit ecc error in L2 Cache failed.\n");
	}

//For getMemorySingleBitErrorCounterDeviceMemory()
    if(getMemorySingleBitErrorCounterDeviceMemory(device, &singleBitErrorDeviceMemoryCount) == 0){
	    printf("the number of single bit ecc error in Device Memory is %lld.\n", singleBitErrorDeviceMemoryCount);
	} else {
	    printf("get single bit ecc error in Device Memory failed.\n");
	}

//For getMemoryDoubleBitErrorCounterDeviceMemory()
    if(getMemoryDoubleBitErrorCounterDeviceMemory(device, &doubleBitErrorDeviceMemoryCount) == 0){
	    printf("the number of double bit ecc error in Device Memory is %lld.\n", doubleBitErrorDeviceMemoryCount);
	} else {
	    printf("get double bit ecc error in Device Memory failed.\n");
	}

//For getMemorySingleBitErrorCounterRegisterFile()
    if(getMemorySingleBitErrorCounterRegisterFile(device, &singleBitErrorRegisterFileCount) == 0){
	    printf("the number of single bit ecc error in Register File is %lld.\n", singleBitErrorRegisterFileCount);
	} else {
	    printf("get single bit ecc error in Register File failed.\n");
	}

//For getMemoryDoubleBitErrorCounterRegisterFile()
    if(getMemoryDoubleBitErrorCounterRegisterFile(device, &doubleBitErrorRegisterFileCount) == 0){
	    printf("the number of double bit ecc error in Register File is %lld.\n", doubleBitErrorRegisterFileCount);
	} else {
	    printf("get double bit ecc error in Register File failed.\n");
	}

//For getMemorySingleBitErrorCounterTextureMemory()
    if(getMemorySingleBitErrorCounterTextureMemory(device, &singleBitErrorTextureMemoryCount) == 0){
	    printf("the number of single bit ecc error in Texture Memory is %lld.\n", singleBitErrorTextureMemoryCount);
	} else {
	    printf("get single bit ecc error in Texture Memory failed.\n");
	}

//For getMemoryDoubleBitErrorCounterTextureMemory()
    if(getMemoryDoubleBitErrorCounterTextureMemory(device, &doubleBitErrorTextureMemoryCount) == 0){
	    printf("the number of double bit ecc error in Texture Memory is %lld.\n", doubleBitErrorTextureMemoryCount);
	} else {
	    printf("get double bit ecc error in Texture Memory failed.\n");
	}

//For getMemorySingleBitErrorCounterTextureShm()
// not support
/*
    if(getMemorySingleBitErrorCounterTextureShm(device, &singleBitErrorTextureShmCount) == 0){
	    printf("the number of single bit ecc error in Shared memory is %lld.\n", singleBitErrorTextureShmCount);
	} else {
	    printf("get single bit ecc error in Shared memory failed.\n");
	}

//For getMemoryDoubleBitErrorCounterTextureShm()
    if(getMemoryDoubleBitErrorCounterTextureShm(device, &doubleBitErrorTextureShmCount) == 0){
	    printf("the number of double bit ecc error in Shared memory is %lld.\n", doubleBitErrorTextureShmCount);
	} else {
	    printf("get double bit ecc error in Shared memory failed.\n");
	}

//For getMemorySingleBitErrorCounterCBU()
    if(getMemorySingleBitErrorCounterCBU(device, &singleBitErrorCBUCount) == 0){
	    printf("the number of single bit ecc error in CBU is %lld.\n", singleBitErrorCBUCount);
	} else {
	    printf("get single bit ecc error in CBU failed.\n");
	}

//For getMemoryDoubleBitErrorCounterCBU()
    if(getMemoryDoubleBitErrorCounterCBU(device, &doubleBitErrorCBUCount) == 0){
	    printf("the number of double bit ecc error in CBU is %lld.\n", doubleBitErrorCBUCount);
	} else {
	    printf("get double bit ecc error in CBU failed.\n");
	}
*/


//int getUtilizationRates()
    if(getUtilizationRates(device, deviceUtilization) == 0){
	    printf("the Utilization of GPU is %u.\n", deviceUtilization->gpu);
	    printf("the Utilization of Memory is %u.\n", deviceUtilization->memory);
	} else {
	    printf("get device's UtilizationRate failed.\n");
	}


//int getEncoderUtilization()




//int getDecoderUtilization()


//int getComputeRunningProcesses()
    if(getComputeRunningProcesses(device, infos) == 0){
	    printf("Process ID is %u.\n", infos->pid);
		printf("used Gpu Memory is %lld.\n",infos->usedGpuMemory);
	} else {
	    printf("get Compute Running Processes failed.\n");
	}



//int getBAR1MemoryInfo()
    if(getBAR1MemoryInfo(device, bar1Memory) == 0){
	    printf("bar1 total memory is %lld.\n", bar1Memory->bar1Total);
		printf("bar1 free memory is %lld.\n", bar1Memory->bar1Free);
		printf("bar1 used memory is %lld.\n", bar1Memory->bar1Used);
	} else {
	    printf("get bar1 memory infos failed.\n");
	}


//int getViolationStatus()

//shutdown
    if(shutdown() == 0){
		printf("nvml shutdown succ.\n");
	} else {
		printf("nvml shutdown err.\n");
		return 1;
	}
	
// free
   free(devicename);
   free(memory);
   free(deviceUtilization);
   free(infos);
   free(bar1Memory);
return 0;	

}



