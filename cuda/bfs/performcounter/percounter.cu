
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <nvml.h>

int init(){
	if(nvmlInit() == NVML_SUCCESS){
		return 0;
	} else {
		return 1;
	}


}

int shutdown(){
	if(nvmlShutdown() == NVML_SUCCESS){
		return 0;
	} else {
		return 1;
	}

}

int handleReturnValue(nvmlReturn_t returnValue){
    if(returnValue == NVML_SUCCESS){
		printf("success.\n");
		return 0;
	} else if(returnValue == NVML_ERROR_UNINITIALIZED){
		printf("error, the library has not been successfully initialized.\n");
		return 1;
	} else if(returnValue == NVML_ERROR_INVALID_ARGUMENT){
		printf("error,device or counter is invalid,or value is NULL.\n");
		return 1;
	} else if(returnValue == NVML_ERROR_NOT_SUPPORTED){
		printf("error,the device does not support this feature.\n");
		return 1;
	} else if(returnValue == NVML_ERROR_GPU_IS_LOST){
		printf("error, the target GPU has fallen off the bus or is otherwise inaccessible.\n");
		return 1;
	} else {
		printf("error unknown.\n");
		return 1;

	}

}

int getDevicehandle(nvmlDevice_t *device){
	unsigned int index = 0;
	if(nvmlDeviceGetHandleByIndex(index,device) == NVML_SUCCESS){
		return 0;
	} else {
		return 1;
	}

}

int getCurrPcieLinkWidth(nvmlDevice_t device, unsigned int *currLinkWidth){
	if(nvmlDeviceGetCurrPcieLinkWidth(device, currLinkWidth) == NVML_SUCCESS) {
		return 0;
	} else {
		return 1;
	}

}


//The device does not support this feature.
/*
int getPcieThroughput(nvmlDevice_t device, unsigned int *value){
	nvmlPcieUtilCounter_t counter1 = NVML_PCIE_UTIL_TX_BYTES;
//  nvmlPcieUtilCounter_t counter2 = NVML_PCIE_UTIL_RX_BYTES;
//  nvmlPcieUtilCounter_t counter3 = NVML_PCIE_UTIL_COUNT;
	nvmlReturn_t returnv = nvmlDeviceGetPcieThroughput(device, counter1, value);
	return handleReturnValue(returnv);

}
*/

int getPcieReplayCounter(nvmlDevice_t device, unsigned int *value){
	nvmlReturn_t returnv = nvmlDeviceGetPcieReplayCounter(device, value);
	return handleReturnValue(returnv);

}

//not support
/*
int getFanSpeed(nvmlDevice_t device, unsigned int *speed){
	nvmlReturn_t returnv = nvmlDeviceGetFanSpeed(device, speed);
	return handleReturnValue(returnv);
}
*/
int getPowerUsage(nvmlDevice_t device, unsigned int *power){
    nvmlReturn_t returnv = nvmlDeviceGetPowerUsage(device, power);
	return handleReturnValue(returnv);

}

//For volta or newer
/*
int getTotalEnergyConsumption(nvmlDevice_t device, unsigned long long *energy){
    nvmlReturn_t returnValue = nvmlDeviceGetTotalEnergyConsumption(device, energy);
	return handleReturnValue(returnValue);
 
}
*/



int getMemoryInfo(nvmlDevice_t device, nvmlMemory_t *memory){
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryInfo(device, memory);
	return handleReturnValue(returnValue);

}



int getTotalSingleBitEccErrors(nvmlDevice_t device, unsigned long long *eccCounts){
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
    nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_CORRECTED;
    nvmlReturn_t returnValue = nvmlDeviceGetTotalEccErrors(device, errorType, counterType, eccCounts);
    return handleReturnValue(returnValue);
	
}



int getTotalDoubleBitEccErrors(nvmlDevice_t device, unsigned long long *eccCounts){
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
    nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_UNCORRECTED;
    nvmlReturn_t returnValue = nvmlDeviceGetTotalEccErrors(device, errorType, counterType, eccCounts);
	return handleReturnValue(returnValue);
}


int getMemorySingleBitErrorCounterL1Cache(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_CORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_L1_CACHE;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
	return handleReturnValue(returnValue);

}


int getMemoryDoubleBitErrorCounterL1Cache(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_UNCORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_L1_CACHE;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemorySingleBitErrorCounterL2Cache(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_CORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_L2_CACHE;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemoryDoubleBitErrorCounterL2Cache(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_UNCORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_L2_CACHE;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemorySingleBitErrorCounterDeviceMemory(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_CORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_DEVICE_MEMORY;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemoryDoubleBitErrorCounterDeviceMemory(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_UNCORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_DEVICE_MEMORY;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemorySingleBitErrorCounterRegisterFile(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_CORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_REGISTER_FILE;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemoryDoubleBitErrorCounterRegisterFile(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_UNCORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_REGISTER_FILE;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemorySingleBitErrorCounterTextureMemory(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_CORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_TEXTURE_MEMORY;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemoryDoubleBitErrorCounterTextureMemory(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_UNCORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_TEXTURE_MEMORY;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemorySingleBitErrorCounterTextureShm(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_CORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_TEXTURE_SHM;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemoryDoubleBitErrorCounterTextureShm(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_UNCORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_TEXTURE_SHM;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemorySingleBitErrorCounterCBU(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_CORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_CBU;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemoryDoubleBitErrorCounterCBU(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_UNCORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_CBU;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getUtilizationRates(nvmlDevice_t device, nvmlUtilization_t *utilization){
    nvmlReturn_t returnValue = nvmlDeviceGetUtilizationRates(device, utilization);
    return handleReturnValue(returnValue);

}


int getEncoderUtilization(nvmlDevice_t device, unsigned int *utilization, unsigned int *samplingPeriodUs){
    nvmlReturn_t returnValue = nvmlDeviceGetEncoderUtilization(device, utilization, samplingPeriodUs);
    return handleReturnValue(returnValue);

}

//Maxwell or newer
/*
int GetEncoderCapacity(nvmlDevice_t device, unsigned int *encoderCapacity){

    //NVML_ENCODER_QUERY_H264 = 0
	//NVML_ENCODER_QUERY_HEVC = 1
    nvmlEncoderType_t encoderQueryType = NVML_ENCODER_QUERY_H264;
    nvmlReturn_t returnValue = nvmlDeviceGetEncoderUtilization(device, encoderQueryType, encoderCapacity);
    return handleReturnValue(returnValue);

}
*/

//Maxwell or newer
/*
int getEncoderStats(nvmlDevice_t device, unsigned int *sessionCount, unsigned int *averageFps, unsigned int *averageLatency){
    nvmlReturn_t returnValue = nvmlDeviceGetEncoderStats(device, sessionCount, averageFps, averageLatency);
    return handleReturnValue(returnValue);

}
*/


int getDecoderUtilization(nvmlDevice_t device, unsigned int *utilization, unsigned int *samplingPeriodUs){

    nvmlReturn_t returnValue = nvmlDeviceGetDecoderUtilization(device, utilization, samplingPeriodUs);
    return handleReturnValue(returnValue);

}

//Maxwell or newer
/*
int getFBCStats(nvmlDevice_t device, nvmlFBCStats_t *fbcStats){
    nvmlReturn_t returnValue = nvmlDeviceGetFBCStats(device, fbcStats);
    return handleReturnValue(returnValue);

}
*/

int getComputeRunningProcesses(nvmlDevice_t device, nvmlProcessInfo_t *infos){
    unsigned int infoCount = 1;
    nvmlReturn_t returnValue = nvmlDeviceGetComputeRunningProcesses(device, &infoCount, infos);
    return handleReturnValue(returnValue);

}


int getBAR1MemoryInfo(nvmlDevice_t device, nvmlBAR1Memory_t *bar1Memory){
    nvmlReturn_t returnValue = nvmlDeviceGetBAR1MemoryInfo(device, bar1Memory);
    return handleReturnValue(returnValue);

}


int getViolationStatus(nvmlDevice_t device, nvmlPerfPolicyType_t perfPolicyType, nvmlViolationTime_t *violTime){
    nvmlReturn_t returnValue = nvmlDeviceGetViolationStatus(device, perfPolicyType, violTime);
    return handleReturnValue(returnValue);

}
//Pascal or newer
/*
int getNvLinkErrorCounterReplay(nvmlDevice_t device, unsigned int link, unsigned long long *counterValue){

    nvmlNvLinkErrorCounter_t counter = NVML_NVLINK_ERROR_DL_REPLAY;
    nvmlReturn_t returnValue = nvmlDeviceGetNvLinkErrorCounter(device, link, counter, counterValue);
    return handleReturnValue(returnValue);
}


int getNvLinkErrorCounterRecovery(nvmlDevice_t device, unsigned int link, unsigned long long *counterValue){
    nvmlNvLinkErrorCounter_t counter = NVML_NVLINK_ERROR_DL_RECOVERY;
    nvmlReturn_t returnValue = nvmlDeviceGetNvLinkErrorCounter(device, link, counter, counterValue);
    return handleReturnValue(returnValue);


}


int getNvLinkErrorCounterCRCFlit(nvmlDevice_t device, unsigned int link, unsigned long long *counterValue){
    nvmlNvLinkErrorCounter_t counter = NVML_NVLINK_ERROR_DL_CRC_FLIT;
    nvmlReturn_t returnValue = nvmlDeviceGetNvLinkErrorCounter(device, link, counter, counterValue);
    return handleReturnValue(returnValue);

}


int getNvLinkErrorCounterCRCData(nvmlDevice_t device, unsigned int link, unsigned long long *counterValue){
    nvmlNvLinkErrorCounter_t counter = NVML_NVLINK_ERROR_DL_CRC_DATA;
    nvmlReturn_t returnValue = nvmlDeviceGetNvLinkErrorCounter(device, link, counter, counterValue);
    return handleReturnValue(returnValue);

}

int getNvLinkUtilizationCounter(nvmlDevice_t device, unsigned int link, unsigned int counter, unsigned long long *rxcounter, unsigned long long *txcounter){
    nvmlReturn_t returnValue = nvmlDeviceGetNvLinkUtilizationCounter(device, link, counter, rxcounter, txcounter);
    return handleReturnValue(returnValue);


}
*/

/*
int getSupportedVgpus(nvmlDevice_t device, unsigned int *vgpuCount, nvmlVgpuTypeId_t *vgpuTypeIds){
    nvmlReturn_t returnValue = nvmlDeviceGetSupportedVgpus(device, vgpuCount, vgpuTypeIds);
    return handleReturnValue(returnValue);
}


int getActiveVgpus(nvmlDevice_t device, unsigned int *vgpuCount, nvmlVgpuInstance_t *vgpuInstances){
    nvmlReturn_t returnValue = nvmlDeviceGetActiveVgpus (device, vgpuCount, vgpuInstances);
    return handleReturnValue(returnValue);

}

int vgpuInstanceGetEncoderCapacity(nvmlVgpuInstance_t vgpuInstance, unsigned int *encoderCapacity){
    nvmlReturn_t returnValue = nvmlVgpuInstanceGetEncoderCapacity(vgpuInstance, encoderCapacity);
    return handleReturnValue(returnValue);

}


int getVgpuUtilization(nvmlDevice_t device, unsigned long long lastSeenTimeStamp, nvmlValueType_t *sampleValType, unsigned int *vgpuInstanceSamplesCount, nvmlVgpuInstanceUtilizationSample_t *utilizationSamples){
    nvmlReturn_t returnValue = nvmlDeviceGetVgpuUtilization(device, lastSeenTimeStamp, sampleValType, vgpuInstanceSamplesCount, utilizationSamples);
    return handleReturnValue(returnValue);
}


int getVgpuProcessUtilization(nvmlDevice_t device, unsigned long long lastSeenTimeStamp, unsigned int *vgpuProcessSamplesCount, nvmlVgpuProcessUtilizationSample_t *utilizationSamples){
    nvmlReturn_t returnValue = nvmlDeviceGetVgpuProcessUtilization(device, lastSeenTimeStamp, vgpuProcessSamplesCount, utilizationSamples);
    return handleReturnValue(returnValue);
}

int vgpuInstanceGetEncoderStats(nvmlVgpuInstance_t vgpuInstance, unsigned int *sessionCount, unsigned int *averageFps, unsigned int *averageLatency){
    nvmlReturn_t returnValue = nvmlVgpuInstanceGetEncoderStats(vgpuInstance, sessionCount, averageFps, averageLatency);
    return handleReturnValue(returnValue);
}


int vgpuInstanceGetFBCStats(nvmlVgpuInstance_t vgpuInstance, nvmlFBCStats_t *fbcStats){
    nvmlReturn_t returnValue = nvmlVgpuInstanceGetFBCStats(vgpuInstance, fbcStats);
    return handleReturnValue(returnValue);

}
*/

//Maxwell or newer
/*
int getProcessUtilization(nvmlDevice_t device, nvmlProcessUtilizationSample_t *utilization, unsigned int *processSamplesCount, unsigned long long lastSeenTimeStamp){
    nvmlReturn_t returnValue = nvmlDeviceGetProcessUtilization(device, utilization, processSamplesCount, lastSeenTimeStamp);
    return handleReturnValue(returnValue);
}
*/

/*
int main(){
	if(init() == 0){
		printf("nvml init succ.\n");
	} else {
		printf("nvml init err.\n");
		return 1;
	}

	unsigned int length = 100;
	char *version =(char*)malloc(length*sizeof(char));
	char *versionnvml = (char*)malloc(length*sizeof(char));
	nvmlSystemGetDriverVersion(version,length);
	printf("the version is %s.\n",version);
	nvmlSystemGetNVMLVersion(versionnvml,length);
	printf("the version of nvml is $s.\n",versionnvml);


	//unsigned int index = 0;
	nvmlDevice_t device;
	char *name = (char*)malloc(sizeof(char)*length);
	unsigned int linkwid;
	if(getDevicehandle(&device) == 0){
		nvmlDeviceGetName(device,name,length);
		printf("the name of device 0 is %s.\n",name);

	}else {
		printf("error.\n");
	}

	if(getCurrPcieLinkWidth(device, &linkwid) == 0){
		printf("current pcie link width is %u.\n",linkwid);
	} else {
		printf("get current pcie link width err.\n");

	}

	unsigned int checksum;
	if(nvmlDeviceGetInforomConfigurationChecksum(device,&checksum)== NVML_SUCCESS){
		printf("checksum is %u.\n",checksum);
	} else {
		printf("get checksum error.\n");
	}

	unsigned int throughput;
	if(getPcieThroughput(device, &throughput) == 0){
		printf("the pcie throughput is %u.\n",throughput);
	} else {
		printf("get pcie throughput error.\n");

	}

	unsigned int pcierepcou;

	if(getPcieReplayCounter(device, &pcierepcou) == 0){
		printf("the pcie replay counter is %u.\n",pcierepcou);
	} else {
		printf("get pcie replay counter error.\n");

	}

	unsigned int speed;
	if(getFanSpeed(device, &speed) == 0){
		printf("the fan speed is %u.\n", speed);
	} else {
		printf("get fan speed error.\n");
	}



	if(shutdown() == 0){
		printf("nvml shutdown succ.\n");
	} else {
		printf("nvml shutdown err.\n");
		return 1;
	}
	

	free(version);
	free(versionnvml);
	free(name);
	return 0;

}
*/
