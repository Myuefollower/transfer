
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <nvml.h>

int init(){
	if(nvmlInit() == NVML_SUCCESS){
		return 0;
	} else {
		return 1;
	}


}

int shutdown(){
	if(nvmlShutdown() == NVML_SUCCESS){
		return 0;
	} else {
		return 1;
	}

}

int handleReturnValue(nvmlReturn_t returnValue){
    if(returnValue == NVML_SUCCESS){
		printf("success.\n");
		return 0;
	} else if(returnValue == NVML_ERROR_UNINITIALIZED){
		printf("error, the library has not been successfully initialized.\n");
		return 1;
	} else if(returnValue == NVML_ERROR_INVALID_ARGUMENT){
		printf("error,device or counter is invalid,or value is NULL.\n");
		return 1;
	} else if(returnValue == NVML_ERROR_NOT_SUPPORTED){
		printf("error,the device does not support this feature.\n");
		return 1;
	} else if(returnValue == NVML_ERROR_GPU_IS_LOST){
		printf("error, the target GPU has fallen off the bus or is otherwise inaccessible.\n");
		return 1;
	} else {
		printf("error unknown.\n");
		return 1;

	}

}

int getDevicehandle(nvmlDevice_t *device){
	unsigned int index = 0;
	if(nvmlDeviceGetHandleByIndex(index,device) == NVML_SUCCESS){
		return 0;
	} else {
		return 1;
	}

}

int getCurrPcieLinkWidth(nvmlDevice_t device, unsigned int *currLinkWidth){
	if(nvmlDeviceGetCurrPcieLinkWidth(device, currLinkWidth) == NVML_SUCCESS) {
		return 0;
	} else {
		return 1;
	}

}


//The device does not support this feature.
/*
int getPcieThroughput(nvmlDevice_t device, unsigned int *value){
	nvmlPcieUtilCounter_t counter1 = NVML_PCIE_UTIL_TX_BYTES;
//  nvmlPcieUtilCounter_t counter2 = NVML_PCIE_UTIL_RX_BYTES;
//  nvmlPcieUtilCounter_t counter3 = NVML_PCIE_UTIL_COUNT;
	nvmlReturn_t returnv = nvmlDeviceGetPcieThroughput(device, counter1, value);
	return handleReturnValue(returnv);

}
*/

int getPcieReplayCounter(nvmlDevice_t device, unsigned int *value){
	nvmlReturn_t returnv = nvmlDeviceGetPcieReplayCounter(device, value);
	return handleReturnValue(returnv);

}

//not support
/*
int getFanSpeed(nvmlDevice_t device, unsigned int *speed){
	nvmlReturn_t returnv = nvmlDeviceGetFanSpeed(device, speed);
	return handleReturnValue(returnv);
}
*/
int getPowerUsage(nvmlDevice_t device, unsigned int *power){
    nvmlReturn_t returnv = nvmlDeviceGetPowerUsage(device, power);
	return handleReturnValue(returnv);

}

//For volta or newer
/*
int getTotalEnergyConsumption(nvmlDevice_t device, unsigned long long *energy){
    nvmlReturn_t returnValue = nvmlDeviceGetTotalEnergyConsumption(device, energy);
	return handleReturnValue(returnValue);
 
}
*/



int getMemoryInfo(nvmlDevice_t device, nvmlMemory_t *memory){
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryInfo(device, memory);
	return handleReturnValue(returnValue);

}



int getTotalSingleBitEccErrors(nvmlDevice_t device, unsigned long long *eccCounts){
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
    nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_CORRECTED;
    nvmlReturn_t returnValue = nvmlDeviceGetTotalEccErrors(device, errorType, counterType, eccCounts);
    return handleReturnValue(returnValue);
	
}



int getTotalDoubleBitEccErrors(nvmlDevice_t device, unsigned long long *eccCounts){
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
    nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_UNCORRECTED;
    nvmlReturn_t returnValue = nvmlDeviceGetTotalEccErrors(device, errorType, counterType, eccCounts);
	return handleReturnValue(returnValue);
}


int getMemorySingleBitErrorCounterL1Cache(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_CORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_L1_CACHE;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
	return handleReturnValue(returnValue);

}


int getMemoryDoubleBitErrorCounterL1Cache(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_UNCORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_L1_CACHE;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemorySingleBitErrorCounterL2Cache(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_CORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_L2_CACHE;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemoryDoubleBitErrorCounterL2Cache(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_UNCORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_L2_CACHE;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemorySingleBitErrorCounterDeviceMemory(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_CORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_DEVICE_MEMORY;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemoryDoubleBitErrorCounterDeviceMemory(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_UNCORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_DEVICE_MEMORY;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemorySingleBitErrorCounterRegisterFile(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_CORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_REGISTER_FILE;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemoryDoubleBitErrorCounterRegisterFile(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_UNCORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_REGISTER_FILE;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemorySingleBitErrorCounterTextureMemory(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_CORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_TEXTURE_MEMORY;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemoryDoubleBitErrorCounterTextureMemory(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_UNCORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_TEXTURE_MEMORY;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemorySingleBitErrorCounterTextureShm(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_CORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_TEXTURE_SHM;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemoryDoubleBitErrorCounterTextureShm(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_UNCORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_TEXTURE_SHM;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemorySingleBitErrorCounterCBU(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_CORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_CBU;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getMemoryDoubleBitErrorCounterCBU(nvmlDevice_t device, unsigned long long *count){

	//NVML_MEMORY_ERROR_TYPE_CORRECTED = 0
	//NVML_MEMORY_ERROR_TYPE_UNCORRECTED = 1
	nvmlMemoryErrorType_t errorType = NVML_MEMORY_ERROR_TYPE_UNCORRECTED;
	
    //NVML_VOLATILE_ECC = 0
	//NVML_AGGREGATE_ECC = 1
    nvmlEccCounterType_t counterType = NVML_VOLATILE_ECC;
	
	nvmlMemoryLocation_t locationType = NVML_MEMORY_LOCATION_CBU;
    nvmlReturn_t returnValue = nvmlDeviceGetMemoryErrorCounter(device, errorType, counterType, locationType, count);
    return handleReturnValue(returnValue);

}


int getUtilizationRates(nvmlDevice_t device, nvmlUtilization_t *utilization){
    nvmlReturn_t returnValue = nvmlDeviceGetUtilizationRates(device, utilization);
    return handleReturnValue(returnValue);

}


int getEncoderUtilization(nvmlDevice_t device, unsigned int *utilization, unsigned int *samplingPeriodUs){
    nvmlReturn_t returnValue = nvmlDeviceGetEncoderUtilization(device, utilization, samplingPeriodUs);
    return handleReturnValue(returnValue);

}

//Maxwell or newer
/*
int GetEncoderCapacity(nvmlDevice_t device, unsigned int *encoderCapacity){

    //NVML_ENCODER_QUERY_H264 = 0
	//NVML_ENCODER_QUERY_HEVC = 1
    nvmlEncoderType_t encoderQueryType = NVML_ENCODER_QUERY_H264;
    nvmlReturn_t returnValue = nvmlDeviceGetEncoderUtilization(device, encoderQueryType, encoderCapacity);
    return handleReturnValue(returnValue);

}
*/

//Maxwell or newer
/*
int getEncoderStats(nvmlDevice_t device, unsigned int *sessionCount, unsigned int *averageFps, unsigned int *averageLatency){
    nvmlReturn_t returnValue = nvmlDeviceGetEncoderStats(device, sessionCount, averageFps, averageLatency);
    return handleReturnValue(returnValue);

}
*/


int getDecoderUtilization(nvmlDevice_t device, unsigned int *utilization, unsigned int *samplingPeriodUs){

    nvmlReturn_t returnValue = nvmlDeviceGetDecoderUtilization(device, utilization, samplingPeriodUs);
    return handleReturnValue(returnValue);

}

//Maxwell or newer
/*
int getFBCStats(nvmlDevice_t device, nvmlFBCStats_t *fbcStats){
    nvmlReturn_t returnValue = nvmlDeviceGetFBCStats(device, fbcStats);
    return handleReturnValue(returnValue);

}
*/

int getComputeRunningProcesses(nvmlDevice_t device, nvmlProcessInfo_t *infos){
    unsigned int infoCount = 1;
    nvmlReturn_t returnValue = nvmlDeviceGetComputeRunningProcesses(device, &infoCount, infos);
    return handleReturnValue(returnValue);

}


int getBAR1MemoryInfo(nvmlDevice_t device, nvmlBAR1Memory_t *bar1Memory){
    nvmlReturn_t returnValue = nvmlDeviceGetBAR1MemoryInfo(device, bar1Memory);
    return handleReturnValue(returnValue);

}


int getViolationStatus(nvmlDevice_t device, nvmlPerfPolicyType_t perfPolicyType, nvmlViolationTime_t *violTime){
    nvmlReturn_t returnValue = nvmlDeviceGetViolationStatus(device, perfPolicyType, violTime);
    return handleReturnValue(returnValue);

}
//Pascal or newer
/*
int getNvLinkErrorCounterReplay(nvmlDevice_t device, unsigned int link, unsigned long long *counterValue){

    nvmlNvLinkErrorCounter_t counter = NVML_NVLINK_ERROR_DL_REPLAY;
    nvmlReturn_t returnValue = nvmlDeviceGetNvLinkErrorCounter(device, link, counter, counterValue);
    return handleReturnValue(returnValue);
}


int getNvLinkErrorCounterRecovery(nvmlDevice_t device, unsigned int link, unsigned long long *counterValue){
    nvmlNvLinkErrorCounter_t counter = NVML_NVLINK_ERROR_DL_RECOVERY;
    nvmlReturn_t returnValue = nvmlDeviceGetNvLinkErrorCounter(device, link, counter, counterValue);
    return handleReturnValue(returnValue);


}


int getNvLinkErrorCounterCRCFlit(nvmlDevice_t device, unsigned int link, unsigned long long *counterValue){
    nvmlNvLinkErrorCounter_t counter = NVML_NVLINK_ERROR_DL_CRC_FLIT;
    nvmlReturn_t returnValue = nvmlDeviceGetNvLinkErrorCounter(device, link, counter, counterValue);
    return handleReturnValue(returnValue);

}


int getNvLinkErrorCounterCRCData(nvmlDevice_t device, unsigned int link, unsigned long long *counterValue){
    nvmlNvLinkErrorCounter_t counter = NVML_NVLINK_ERROR_DL_CRC_DATA;
    nvmlReturn_t returnValue = nvmlDeviceGetNvLinkErrorCounter(device, link, counter, counterValue);
    return handleReturnValue(returnValue);

}

int getNvLinkUtilizationCounter(nvmlDevice_t device, unsigned int link, unsigned int counter, unsigned long long *rxcounter, unsigned long long *txcounter){
    nvmlReturn_t returnValue = nvmlDeviceGetNvLinkUtilizationCounter(device, link, counter, rxcounter, txcounter);
    return handleReturnValue(returnValue);


}
*/

/*
int getSupportedVgpus(nvmlDevice_t device, unsigned int *vgpuCount, nvmlVgpuTypeId_t *vgpuTypeIds){
    nvmlReturn_t returnValue = nvmlDeviceGetSupportedVgpus(device, vgpuCount, vgpuTypeIds);
    return handleReturnValue(returnValue);
}


int getActiveVgpus(nvmlDevice_t device, unsigned int *vgpuCount, nvmlVgpuInstance_t *vgpuInstances){
    nvmlReturn_t returnValue = nvmlDeviceGetActiveVgpus (device, vgpuCount, vgpuInstances);
    return handleReturnValue(returnValue);

}

int vgpuInstanceGetEncoderCapacity(nvmlVgpuInstance_t vgpuInstance, unsigned int *encoderCapacity){
    nvmlReturn_t returnValue = nvmlVgpuInstanceGetEncoderCapacity(vgpuInstance, encoderCapacity);
    return handleReturnValue(returnValue);

}


int getVgpuUtilization(nvmlDevice_t device, unsigned long long lastSeenTimeStamp, nvmlValueType_t *sampleValType, unsigned int *vgpuInstanceSamplesCount, nvmlVgpuInstanceUtilizationSample_t *utilizationSamples){
    nvmlReturn_t returnValue = nvmlDeviceGetVgpuUtilization(device, lastSeenTimeStamp, sampleValType, vgpuInstanceSamplesCount, utilizationSamples);
    return handleReturnValue(returnValue);
}


int getVgpuProcessUtilization(nvmlDevice_t device, unsigned long long lastSeenTimeStamp, unsigned int *vgpuProcessSamplesCount, nvmlVgpuProcessUtilizationSample_t *utilizationSamples){
    nvmlReturn_t returnValue = nvmlDeviceGetVgpuProcessUtilization(device, lastSeenTimeStamp, vgpuProcessSamplesCount, utilizationSamples);
    return handleReturnValue(returnValue);
}

int vgpuInstanceGetEncoderStats(nvmlVgpuInstance_t vgpuInstance, unsigned int *sessionCount, unsigned int *averageFps, unsigned int *averageLatency){
    nvmlReturn_t returnValue = nvmlVgpuInstanceGetEncoderStats(vgpuInstance, sessionCount, averageFps, averageLatency);
    return handleReturnValue(returnValue);
}


int vgpuInstanceGetFBCStats(nvmlVgpuInstance_t vgpuInstance, nvmlFBCStats_t *fbcStats){
    nvmlReturn_t returnValue = nvmlVgpuInstanceGetFBCStats(vgpuInstance, fbcStats);
    return handleReturnValue(returnValue);

}
*/

//Maxwell or newer
/*
int getProcessUtilization(nvmlDevice_t device, nvmlProcessUtilizationSample_t *utilization, unsigned int *processSamplesCount, unsigned long long lastSeenTimeStamp){
    nvmlReturn_t returnValue = nvmlDeviceGetProcessUtilization(device, utilization, processSamplesCount, lastSeenTimeStamp);
    return handleReturnValue(returnValue);
}
*/





void collectNvmlData(FILE *datafp){

	//datafp = fopen("nvmlData.txt", "a+");


//init
/*
    if(init() == 0){
		fprintf(datafp, "%s\n", "nvml init succ.\n");
	} else {
		fprintf(datafp, "%s\n", "nvml init err.\n");
	}
*/
    
//For Variable and malloc
	unsigned int length = 100;
        nvmlDevice_t device;
	
	char *devicename = (char*)malloc(sizeof(char)*length);
	
	unsigned int linkwid;

       //unsigned int pciethroughput;	
	
	unsigned int pcieReplayCount;
	
	//unsigned int fanspeed;
	
	unsigned int powerusage;
	
	unsigned long long eccCountsTotalSingle;
	
	unsigned long long eccCountsTotalDouble;
	
//	unsigned long long energyConsumption;
	
	nvmlMemory_t *memory = (nvmlMemory_t*)malloc(sizeof(nvmlMemory_t));
	
	unsigned long long singleBitErrorL1CacheCount;
	
	unsigned long long doubleBitErrorL1CacheCount;
	
	unsigned long long singleBitErrorL2CacheCount;
	
	unsigned long long doubleBitErrorL2CacheCount;
	
	unsigned long long singleBitErrorDeviceMemoryCount;
	
	unsigned long long doubleBitErrorDeviceMemoryCount;
	
	unsigned long long singleBitErrorRegisterFileCount;
	
	unsigned long long doubleBitErrorRegisterFileCount;
	
	unsigned long long singleBitErrorTextureMemoryCount;
	
	unsigned long long doubleBitErrorTextureMemoryCount;
	
	//unsigned long long singleBitErrorTextureShmCount;
	
	//unsigned long long doubleBitErrorTextureShmCount;
	
	//unsigned long long singleBitErrorCBUCount;
	
	//unsigned long long doubleBitErrorCBUCount;
	
	nvmlUtilization_t *deviceUtilization = (nvmlUtilization_t*)malloc(sizeof(nvmlUtilization_t));
	
	nvmlProcessInfo_t *infos = (nvmlProcessInfo_t*)malloc(sizeof(nvmlProcessInfo_t));
	
	nvmlBAR1Memory_t *bar1Memory = (nvmlBAR1Memory_t*)malloc(sizeof(nvmlBAR1Memory_t));
	


// For Test
   // fprintf(datafp, "%s\n", "nvml init err.\n");
   // For Example
    fprintf(datafp, "%s %d.\n", "the clock count is", NVML_CLOCK_COUNT);
    fprintf(datafp, "%s %d.\n", "the graphics clock is", NVML_CLOCK_GRAPHICS);
    fprintf(datafp, "%s %d.\n", "the SM clock is", NVML_CLOCK_SM);
    fprintf(datafp, "%s %d.\n", "the Memory clock is", NVML_CLOCK_MEM);
   // printf("the Video clock is %d.\n",NVML_CLOCK_VIDEO);    
//  For getDevicehandle()

    if (getDevicehandle(&device) == 0){
        fprintf(datafp, "%s.\n", "get device handle success");
		nvmlDeviceGetName(device,devicename,length);
		fprintf(datafp, "%s %s.\n", "the name of device 0 is", devicename);
	} else {
	    fprintf(datafp, "%s.\n", "get device handle error");
	    //return 1;
	}

//  For getCurrPcieLinkWidth()

    if(getCurrPcieLinkWidth(device, &linkwid) == 0){
		fprintf(datafp, "%s %u.\n", "current pcie link width is", linkwid);
	} else {
		fprintf(datafp, "%s.\n", "get current pcie link width err");
	}

//  For getPcieThroughput()
 //not support
 /*   
	if(getPcieThroughput(device, &pciethroughput) == 0){
		fprintf(datafp, "%s %u.\n", "the pcie throughput is", pciethroughput);
	} else {
		fprintf(datafp, "%s.\n", "get pcie throughput error");

	}
*/
//For getPcieReplayCounter()

    if(getPcieReplayCounter(device, &pcieReplayCount) == 0){
	    fprintf(datafp, "%s %u.\n", "the pcie replay counter is", pcieReplayCount);
	} else {
	    fprintf(datafp, "%s.\n", "get pcie replay counter error");
	}

//For getFanSpeed()
//not support
/*
    if(getFanSpeed(device, &fanspeed) == 0){
		fprintf("the fan speed is %u.\n", fanspeed);
	} else {
		fprintf("get fan speed error.\n");
	}
*/
//For getPowerUsage()

    if(getPowerUsage(device, &powerusage) == 0){
	    fprintf(datafp, "%s %u.\n", "the power usage is", powerusage);
	} else {
	    fprintf(datafp, "%s.\n", "get power usage error");
	}

//For getTotalEnergyConsumption()
//volta or newer
/*
    if(getTotalEnergyConsumption(device, &energyConsumption) == 0){
	    fprintf(datafp, "%s %lld.\n", "the total energy consumption is", energyConsumption);
	} else {
	    fprintf(datafp, "%s.\n", "get total energy consumption error");
	}
*/

//For getMemoryInfo()
    if(getMemoryInfo(device, memory) == 0){
	    fprintf(datafp, "%s %lld.\n", "the total memory is", memory->total);
		fprintf(datafp, "%s %lld.\n", "the free memory is", memory->free);
		fprintf(datafp, "%s %lld.\n", "the used memory is", memory->used);
    
    } else {
	    fprintf(datafp, "%s.\n", "get memory info error");
	}


//For getTotalSingleBitEccErrors()
    if(getTotalSingleBitEccErrors(device, &eccCountsTotalSingle) == 0){
	    fprintf(datafp, "%s %lld.\n", "the number of total single bit ecc error is", eccCountsTotalSingle);
	} else {
	    fprintf(datafp, "%s.\n", "get total single bit ecc error failed");
	}



//For getTotalDoubleBitEccErrors()
    if(getTotalDoubleBitEccErrors(device, &eccCountsTotalDouble) == 0){
	    fprintf(datafp, "%s %lld.\n", "the number of total double bit ecc error is", eccCountsTotalDouble);
	} else {
	    fprintf(datafp, "%s.\n", "get total double bit ecc error failed");
	}



//For getMemorySingleBitErrorCounterL1Cache()
    if(getMemorySingleBitErrorCounterL1Cache(device, &singleBitErrorL1CacheCount) == 0){
	    fprintf(datafp, "%s %lld.\n", "the number of single bit ecc error in L1 Cache is", singleBitErrorL1CacheCount);
		
	} else {
	    fprintf(datafp, "%s.\n", "get single bit ecc error in L1 Cache failed");
	}


//For getMemoryDoubleBitErrorCounterL1Cache()
    if(getMemoryDoubleBitErrorCounterL1Cache(device, &doubleBitErrorL1CacheCount) == 0){
	    fprintf(datafp, "%s %lld.\n", "the number of double bit ecc error in L1 Cache is", eccCountsTotalDouble);
	} else {
	    fprintf(datafp, "%s.\n", "get double bit ecc error in L1 Cache failed");
	}

//For getMemorySingleBitErrorCounterL2Cache()
    if(getMemorySingleBitErrorCounterL2Cache(device, &singleBitErrorL2CacheCount) == 0){
	    fprintf(datafp, "%s %lld.\n", "the number of single bit ecc error in L2 Cache is", singleBitErrorL2CacheCount);
	} else {
	    fprintf(datafp, "%s.\n", "get single bit ecc error in L2 Cache failed");
	}

//For getMemoryDoubleBitErrorCounterL2Cache()
    if(getMemoryDoubleBitErrorCounterL2Cache(device, &doubleBitErrorL2CacheCount) == 0){
	    fprintf(datafp, "%s %lld.\n", "the number of double bit ecc error in L2 Cache is", doubleBitErrorL2CacheCount);
	} else {
	    fprintf(datafp, "%s.\n", "get double bit ecc error in L2 Cache failed");
	}

//For getMemorySingleBitErrorCounterDeviceMemory()
    if(getMemorySingleBitErrorCounterDeviceMemory(device, &singleBitErrorDeviceMemoryCount) == 0){
	    fprintf(datafp, "%s %lld.\n", "the number of single bit ecc error in Device Memory is", singleBitErrorDeviceMemoryCount);
	} else {
	    fprintf(datafp, "%s.\n", "get single bit ecc error in Device Memory failed");
	}

//For getMemoryDoubleBitErrorCounterDeviceMemory()
    if(getMemoryDoubleBitErrorCounterDeviceMemory(device, &doubleBitErrorDeviceMemoryCount) == 0){
	    fprintf(datafp, "%s %lld.\n", "the number of double bit ecc error in Device Memory is", doubleBitErrorDeviceMemoryCount);
	} else {
	    fprintf(datafp, "%s.\n", "get double bit ecc error in Device Memory failed");
	}

//For getMemorySingleBitErrorCounterRegisterFile()
    if(getMemorySingleBitErrorCounterRegisterFile(device, &singleBitErrorRegisterFileCount) == 0){
	    fprintf(datafp, "%s %lld.\n", "the number of single bit ecc error in Register File is", singleBitErrorRegisterFileCount);
	} else {
	    fprintf(datafp, "%s.\n", "get single bit ecc error in Register File failed");
	}

//For getMemoryDoubleBitErrorCounterRegisterFile()
    if(getMemoryDoubleBitErrorCounterRegisterFile(device, &doubleBitErrorRegisterFileCount) == 0){
	    fprintf(datafp, "%s %lld.\n", "the number of double bit ecc error in Register File is", doubleBitErrorRegisterFileCount);
	} else {
	    fprintf(datafp, "%s.\n", "get double bit ecc error in Register File failed");
	}

//For getMemorySingleBitErrorCounterTextureMemory()
    if(getMemorySingleBitErrorCounterTextureMemory(device, &singleBitErrorTextureMemoryCount) == 0){
	    fprintf(datafp, "%s %lld.\n", "the number of single bit ecc error in Texture Memory is", singleBitErrorTextureMemoryCount);
	} else {
	    fprintf(datafp, "%s.\n", "get single bit ecc error in Texture Memory failed");
	}

//For getMemoryDoubleBitErrorCounterTextureMemory()
    if(getMemoryDoubleBitErrorCounterTextureMemory(device, &doubleBitErrorTextureMemoryCount) == 0){
	    fprintf(datafp, "%s %lld.\n", "the number of double bit ecc error in Texture Memory is", doubleBitErrorTextureMemoryCount);
	} else {
	    fprintf(datafp, "%s.\n", "get double bit ecc error in Texture Memory failed");
	}

//For getMemorySingleBitErrorCounterTextureShm()
// not support
/*
    if(getMemorySingleBitErrorCounterTextureShm(device, &singleBitErrorTextureShmCount) == 0){
	    fprintf("the number of single bit ecc error in Shared memory is %lld.\n", singleBitErrorTextureShmCount);
	} else {
	    fprintf("get single bit ecc error in Shared memory failed.\n");
	}

//For getMemoryDoubleBitErrorCounterTextureShm()
    if(getMemoryDoubleBitErrorCounterTextureShm(device, &doubleBitErrorTextureShmCount) == 0){
	    fprintf("the number of double bit ecc error in Shared memory is %lld.\n", doubleBitErrorTextureShmCount);
	} else {
	    fprintf("get double bit ecc error in Shared memory failed.\n");
	}

//For getMemorySingleBitErrorCounterCBU()
    if(getMemorySingleBitErrorCounterCBU(device, &singleBitErrorCBUCount) == 0){
	    fprintf("the number of single bit ecc error in CBU is %lld.\n", singleBitErrorCBUCount);
	} else {
	    fprintf("get single bit ecc error in CBU failed.\n");
	}

//For getMemoryDoubleBitErrorCounterCBU()
    if(getMemoryDoubleBitErrorCounterCBU(device, &doubleBitErrorCBUCount) == 0){
	    fprintf("the number of double bit ecc error in CBU is %lld.\n", doubleBitErrorCBUCount);
	} else {
	    fprintf("get double bit ecc error in CBU failed.\n");
	}
*/


//int getUtilizationRates()
    if(getUtilizationRates(device, deviceUtilization) == 0){
	    fprintf(datafp, "%s %u.\n", "the Utilization of GPU is", deviceUtilization->gpu);
	    fprintf(datafp, "%s %u.\n", "the Utilization of Memory is", deviceUtilization->memory);
	} else {
	    fprintf(datafp, "%s.\n", "get device's UtilizationRate failed");
	}


//int getEncoderUtilization()




//int getDecoderUtilization()


//int getComputeRunningProcesses()
    if(getComputeRunningProcesses(device, infos) == 0){
	    fprintf(datafp, "%s %u.\n", "Process ID is", infos->pid);
		fprintf(datafp, "%s %lld.\n", "used Gpu Memory is",infos->usedGpuMemory);
	} else {
	    fprintf(datafp, "%s.\n", "get Compute Running Processes failed");
	}



//int getBAR1MemoryInfo()
    if(getBAR1MemoryInfo(device, bar1Memory) == 0){
	    fprintf(datafp, "%s %lld.\n", "bar1 total memory is", bar1Memory->bar1Total);
		fprintf(datafp, "%s %lld.\n", "bar1 free memory is", bar1Memory->bar1Free);
		fprintf(datafp, "%s %lld.\n", "bar1 used memory is", bar1Memory->bar1Used);
	} else {
	    fprintf(datafp, "%s.\n", "get bar1 memory infos failed");
	}


    fprintf(datafp, "%s.\n", "####################");
    fprintf(datafp, "%s.\n", "####################");
	
//int getViolationStatus()

//shutdown
/*
    if(shutdown() == 0){
		printf("nvml shutdown succ.\n");
	} else {
		printf("nvml shutdown err.\n");
		return 1;
	}
*/	
// free

   free(devicename);
   free(memory);
   free(deviceUtilization);
   free(infos);
   free(bar1Memory);
//
    //fclose(datafp);
}

